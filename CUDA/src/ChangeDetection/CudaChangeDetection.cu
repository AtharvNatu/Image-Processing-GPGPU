#include "hip/hip_runtime.h"
#include "../../include/ChangeDetection/CudaChangeDetection.cuh"

// CUDA Kernel Definitions
__global__ void cudaChangeDetection(uchar3 *oldImage, uchar3 *newImage, uchar3 *outputImage, int threshold, int size, int grayscale)
{
    // Variable Declarations
    uchar_t oldGreyValue, newGreyValue, difference;

    // Code
    int pixelId = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (pixelId < size)
    {
        oldGreyValue = (uchar_t)(
                        (0.299 * (uchar_t)oldImage[pixelId].x) +
                        (0.587 * (uchar_t)oldImage[pixelId].y) +
                        (0.114 * (uchar_t)oldImage[pixelId].z)
                    );

        newGreyValue = (uchar_t)(
                        (0.299 * (uchar_t)newImage[pixelId].x) +
                        (0.587 * (uchar_t)newImage[pixelId].y) +
                        (0.114 * (uchar_t)newImage[pixelId].z)
                    );

        difference = abs(oldGreyValue - newGreyValue);

        if (grayscale)
        {
            if (difference >= threshold)
            {
                outputImage[pixelId].x = 255;
                outputImage[pixelId].y = 0;
                outputImage[pixelId].z = 0;
            }
            else
            {
                outputImage[pixelId].x = oldGreyValue;
                outputImage[pixelId].y = oldGreyValue;
                outputImage[pixelId].z = oldGreyValue;
            }
        }
        else
        {
            if (difference >= threshold)
            {
                outputImage[pixelId].x = 255;
                outputImage[pixelId].y = 255;
                outputImage[pixelId].z = 255;
            }
        }
    }
}

// Member Function Definitions

//* DEBUG Mode
CudaChangeDetection::CudaChangeDetection(void)
{
    // Code
    imageUtils = new ImageUtils();
    cudaUtils = new CudaUtils();
    otsuThreshold = new OtsuThresholdCuda();
}

//* RELEASE Mode
CudaChangeDetection::CudaChangeDetection(std::string logFilePath)
{
    // Code
    logger = Logger::getInstance(logFilePath);
    cudaUtils = new CudaUtils();
    imageUtils = new ImageUtils();
    otsuThreshold = new OtsuThresholdCuda();
}

double CudaChangeDetection::detectChanges(std::string oldImagePath, std::string newImagePath, std::string outputPath, bool grayscale)
{
    // Variable Declarations
    cv::String outputImagePath;
    std::string outputFileName;
    double gpuTime = 0;

    // Code

    //* Check Validity of Input Images
    if (!std::filesystem::exists(oldImagePath) || !std::filesystem::exists(newImagePath))
    {
        #if RELEASE
            logger->printLog("Error : Invalid Input Image ... Exiting !!!");
        #else
            std::cerr << std::endl << "Error : Invalid Input Image ... Exiting !!!" << std::endl;
        #endif

        exit(FILE_ERROR);
    }

    // Input and Output File
    std::filesystem::path oldFilePath = std::filesystem::path(oldImagePath).stem();
    std::filesystem::path newFilePath = std::filesystem::path(newImagePath).stem();

    if (grayscale)
        outputFileName = oldFilePath.string() + ("_" + newFilePath.string()) + ("_Changes_Grayscale_CUDA" + std::filesystem::path(oldImagePath).extension().string());
    else
        outputFileName = oldFilePath.string() + ("_" + newFilePath.string()) + ("_Changes_Binary_CUDA" + std::filesystem::path(oldImagePath).extension().string());
    
    #if (OS == 1)
        outputImagePath = outputPath + ("\\" + outputFileName);
    #elif (OS == 2 || OS == 3)
        outputImagePath = outputPath + ("/" + outputFileName);
    #endif

    // Load Images
    cv::Mat oldImage = imageUtils->loadImage(oldImagePath);
    cv::Mat newImage = imageUtils->loadImage(newImagePath);

    //* 1. Preprocessing
    if (oldImage.cols != newImage.cols || oldImage.rows != newImage.rows)
    {
        #if RELEASE
            logger->printLog("Error : Invalid Spatial Resolution ... Input Images With Same Resolution ... Exiting !!!");     
        #else
            std::cerr << std::endl << "Error : Invalid Spatial Resolution ... Input Images With Same Resolution ... Exiting !!!" << std::endl;
        #endif

        newImage.release();
        oldImage.release();

        exit(FILE_ERROR);
    }

    //* Empty Output Image => CV_8UC3 = 3-channel RGB Image
    cv::Mat outputImage(oldImage.rows, oldImage.cols, CV_8UC3, cv::Scalar(0, 0, 0));

    size_t size = oldImage.size().height * oldImage.size().width;

    //* 2. Ostu Thresholding
    int threshold1 = otsuThreshold->computeThreshold(&oldImage, &gpuTime, imageUtils, cudaUtils);
    int threshold2 = otsuThreshold->computeThreshold(&newImage, &gpuTime, imageUtils, cudaUtils);
    int meanThreshold = (threshold1 + threshold2) / 2;

    hostOldImage = new uchar3[size];
    hostNewImage = new uchar3[size];
    hostOutputImage = new uchar3[size];

    cudaUtils->convertImageToPixelArr(oldImage.data, hostOldImage, size);
    cudaUtils->convertImageToPixelArr(newImage.data, hostNewImage, size);
    
    cudaUtils->memAlloc((void**)&deviceOldImage, size * sizeof(uchar3));
    cudaUtils->memAlloc((void**)&deviceNewImage, size * sizeof(uchar3));
    cudaUtils->memAlloc((void**)&deviceOutputImage, size * sizeof(uchar3));

    cudaUtils->memCopy(deviceOldImage, hostOldImage, size * sizeof(uchar3), hipMemcpyHostToDevice);
    cudaUtils->memCopy(deviceNewImage, hostNewImage, size * sizeof(uchar3), hipMemcpyHostToDevice);

    //* CUDA Kernel Configuration
    dim3 BLOCKS((size + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK);

    //* 3. Differencing
    cudaUtils->createEvent(&start);
    cudaUtils->createEvent(&end);
    cudaUtils->recordEvent(start, 0);
    {   
        if (grayscale)
            cudaChangeDetection<<<BLOCKS, THREADS_PER_BLOCK>>>(deviceOldImage, deviceNewImage, deviceOutputImage, meanThreshold, size, 1);
        else
            cudaChangeDetection<<<BLOCKS, THREADS_PER_BLOCK>>>(deviceOldImage, deviceNewImage, deviceOutputImage, meanThreshold, size, 0);
    }
    cudaUtils->recordEvent(end, 0);
    cudaUtils->syncEvent(end);
    cudaUtils->getEventElapsedTime(&gpuTime, start, end);
    gpuTime /= 1000.0;  //* Milliseconds to Seconds

    cudaUtils->memCopy(hostOutputImage, deviceOutputImage, size * sizeof(uchar3), hipMemcpyDeviceToHost);
    cudaUtils->convertPixelArrToImage(hostOutputImage, outputImage.data, size);
    
    //* Save Image
    imageUtils->saveImage(outputImagePath, &outputImage);

    cleanup();

    outputImage.release();
    newImage.release();
    oldImage.release();

    return gpuTime;
}

void CudaChangeDetection::cleanup(void)
{
    //* Cleanup Code
    cudaUtils->destroyEvent(end);
    cudaUtils->destroyEvent(start);

    cudaUtils->memFree((void**)&deviceOutputImage);
    cudaUtils->memFree((void**)&deviceNewImage);
    cudaUtils->memFree((void**)&deviceOldImage);

    delete[] hostOutputImage;
    hostOutputImage = nullptr;

    delete[] hostNewImage;
    hostNewImage = nullptr;

    delete[] hostOldImage;
    hostOldImage = nullptr;
}

CudaChangeDetection::~CudaChangeDetection()
{
    // Code
    delete otsuThreshold;
    otsuThreshold = nullptr;

    delete cudaUtils;
    cudaUtils = nullptr;
    
    delete imageUtils;
    imageUtils = nullptr;

    #if RELEASE
        logger->deleteInstance();
    #endif
}
