#include "hip/hip_runtime.h"
#include "../../include/Common/CudaUtils.cuh"

// Function Definitions
void CudaUtils::memAlloc(void **devPtr, size_t size)
{
    // Code
    hipError_t result = hipMalloc(devPtr, size);
    if (result != hipSuccess)
    {
        #if RELEASE
            Logger *logger = Logger::getInstance("IPUG.log");
            logger->printLog("Error : Failed To Allocate Memory On GPU : %s", hipGetErrorString(result), " ... Exiting !!!");
            logger->deleteInstance();
        #else
            std::cerr << std::endl << "Error : Failed To Allocate Memory On GPU : " << hipGetErrorString(result) << " ... Exiting !!!" << std::endl;
        #endif

        exit(CUDA_ERROR);
    }
}

void CudaUtils::memSet(void *devPtr, int value, size_t count)
{
    // Code
    hipError_t result = hipMemset(devPtr, value, count);
    if (result != hipSuccess)
    {
        #if RELEASE
            Logger *logger = Logger::getInstance("IPUG.log");
            logger->printLog("Error : Failed To Initialize Memory On GPU : %s", hipGetErrorString(result), " ... Exiting !!!");
            logger->deleteInstance();
        #else
            std::cerr << std::endl << "Error : Failed To Initialize Memory On GPU : " << hipGetErrorString(result) << " ... Exiting !!!" << std::endl;
        #endif

        exit(CUDA_ERROR);
    }
}

void CudaUtils::memCopy(void *dst, const void *src, size_t count, hipMemcpyKind kind)
{
    // Code
    hipError_t result = hipMemcpy(dst, src, count, kind);
    if (result != hipSuccess)
    {
        if (kind == hipMemcpyHostToDevice)
        {
            #if RELEASE
                Logger *logger = Logger::getInstance("IPUG.log");
                logger->printLog("Error : Failed To Copy Memory From CPU To GPU : %s", hipGetErrorString(result), " ... Exiting !!!");
                logger->deleteInstance();
            #else
                std::cerr << std::endl << "Error : Failed To Copy Memory From CPU To GPU : " << hipGetErrorString(result) << " ... Exiting !!!" << std::endl;
            #endif
        }
        else if (kind == hipMemcpyDeviceToHost)
        {
            #if RELEASE
                Logger *logger = Logger::getInstance("IPUG.log");
                logger->printLog("Error : Failed To Copy Memory From GPU To CPU : %s", hipGetErrorString(result), " ... Exiting !!!");
                logger->deleteInstance();
            #else
                std::cerr << std::endl << "Error : Failed To Copy Memory From GPU To CPU : " << hipGetErrorString(result) << " ... Exiting !!!" << std::endl;
            #endif
        }
       
        exit(CUDA_ERROR);
    }
}

void CudaUtils::memFree(void **devPtr)
{
    // Code
    if (*devPtr)
    {
        if (hipFree(*devPtr) == hipSuccess);
            *devPtr = nullptr;
    }
}

void CudaUtils::createEvent(hipEvent_t *event)
{
    // Code
    hipError_t result = hipEventCreate(event);
    if (result != hipSuccess)
    {
        #if RELEASE
            Logger *logger = Logger::getInstance("IPUG.log");
            logger->printLog("Error : Failed To Create Event : %s", hipGetErrorString(result), " ... Exiting !!!");
            logger->deleteInstance();
        #else
            std::cerr << std::endl << "Error : Failed To Create Event : " << hipGetErrorString(result) << " ... Exiting !!!" << std::endl;
        #endif

        exit(CUDA_ERROR);
    }
}

void CudaUtils::recordEvent(hipEvent_t event, hipStream_t stream)
{
    // Code
    hipError_t result = hipEventRecord(event, stream);
    if (result != hipSuccess)
    {
        #if RELEASE
            Logger *logger = Logger::getInstance("IPUG.log");
            logger->printLog("Error : Failed To Record Event : %s", hipGetErrorString(result), " ... Exiting !!!");
            logger->deleteInstance();
        #else
            std::cerr << std::endl << "Error : Failed To Record Event : " << hipGetErrorString(result) << " ... Exiting !!!" << std::endl;
        #endif

        exit(CUDA_ERROR);
    }
}

void CudaUtils::syncEvent(hipEvent_t event)
{
    // Code
    hipError_t result = hipEventSynchronize(event);
    if (result != hipSuccess)
    {
        #if RELEASE
            Logger *logger = Logger::getInstance("IPUG.log");
            logger->printLog("Error : Failed To Synchronize Event : %s", hipGetErrorString(result), " ... Exiting !!!");
            logger->deleteInstance();
        #else
            std::cerr << std::endl << "Error : Failed To Synchronize Event : " << hipGetErrorString(result) << " ... Exiting !!!" << std::endl;
        #endif

        exit(CUDA_ERROR);
    }
}

void CudaUtils::getEventElapsedTime(double *ms, hipEvent_t start, hipEvent_t end)
{
    // Code
    float elapsedTime = 0.0F;
    hipError_t result = hipEventElapsedTime(&elapsedTime, start, end);
    if (result != hipSuccess)
    {
        #if RELEASE
            Logger *logger = Logger::getInstance("IPUG.log");
            logger->printLog("Error : Failed To Get Event Elapsed Time : %s", hipGetErrorString(result), " ... Exiting !!!");
            logger->deleteInstance();
        #else
            std::cerr << std::endl << "Error : Failed To Get Event Elapsed Time : " << hipGetErrorString(result) << " ... Exiting !!!" << std::endl;
        #endif

        exit(CUDA_ERROR);
    }
    *ms += elapsedTime;
}

void CudaUtils::destroyEvent(hipEvent_t event)
{
    // Code
    hipError_t result = hipEventDestroy(event);
    if (result != hipSuccess)
    {
        #if RELEASE
            Logger *logger = Logger::getInstance("IPUG.log");
            logger->printLog("Error : Failed To Synchronize Event : %s", hipGetErrorString(result), " ... Exiting !!!");
            logger->deleteInstance();
        #else
            std::cerr << std::endl << "Error : Failed To Synchronize Event : " << hipGetErrorString(result) << " ... Exiting !!!" << std::endl;
        #endif

        exit(CUDA_ERROR);
    }
}

void CudaUtils::convertImageToPixelArr(uchar_t *imageData, uchar3 *pixelArray, size_t size)
{
    // Code
    for (size_t i = 0; i < size; i++, imageData += 3)
    {
        pixelArray[i].x = imageData[2];
        pixelArray[i].y = imageData[1];
        pixelArray[i].z = imageData[0];
    }
}

void CudaUtils::convertPixelArrToImage(uchar3 *pixelArray, uchar_t *imageData, size_t size)
{
    // Code
    for (size_t i = 0; i < size; i++, imageData += 3)
    {
        imageData[2] = pixelArray[i].x;
        imageData[1] = pixelArray[i].y;
        imageData[0] = pixelArray[i].z;
    }
}
