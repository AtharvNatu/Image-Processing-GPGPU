#include "hip/hip_runtime.h"
#include "../../include/Common/CudaUtils.cuh"

// Function Definitions
void cudaMemAlloc(void **devPtr, size_t size)
{
    // Code
    hipError_t result = hipMalloc(devPtr, size);
    if (result != hipSuccess)
    {
        #if RELEASE
            Logger *logger = Logger::getInstance("IPUG.log");
            logger->printLog("Error : Failed To Allocate Memory On GPU : %s", hipGetErrorString(result), " ... Exiting !!!");
            logger->deleteInstance();
        #else
            std::cerr << std::endl << "Error : Failed To Allocate Memory On GPU : " << hipGetErrorString(result) << " ... Exiting !!!" << std::endl;
        #endif

        exit(CUDA_ERROR);
    }
}

void cudaMemCopy(void *dst, const void *src, size_t count, hipMemcpyKind kind)
{
    // Code
    hipError_t result = hipMemcpy(dst, src, count, kind);
    if (result != hipSuccess)
    {
        #if RELEASE
            Logger *logger = Logger::getInstance("IPUG.log");
            logger->printLog("Error : Failed To Copy Memory On GPU : %s", hipGetErrorString(result), " ... Exiting !!!");
            logger->deleteInstance();
        #else
            std::cerr << std::endl << "Error : Failed To Copy Memory From : " << src << " To " << dst << " : " << hipGetErrorString(result) << " ... Exiting !!!" << std::endl;
        #endif

        exit(CUDA_ERROR);
    }
}

void cudaMemFree(void **devPtr)
{
    // Code
    if (*devPtr)
    {
        hipFree(*devPtr);
        *devPtr = NULL;
    }
}
